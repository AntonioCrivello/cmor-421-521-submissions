#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Define size of halo
#define HALO 1

__global__ void stencil(const int N, float *y, const float *x) {

    // Define shared memory with halo points include
    __shared__ float s_x[blockSize + 2];

    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    const int tid = threadIdx.x ;

    if (i < N) {
      s_x[tid + HALO] = x[i];

      s_x[tid] = (tid < HALO) ? x[i - HALO] : x[i];
      s_x[tid + blockDim. + HALO] ? x[i + blockDim.x + HALO] : x[i];

      __syncthreads();

      

    }





    // s_x = 0.f;
    // s_x[tid] = (tid < HALO) ? x[i - HALO] : x[i];
    // s_x[tid + blockSize] = (tid < HALO) ? x[i + blockSize + HALO]: x[i];

    // s_xn1 = (i == 0 && tid == 0) ? s_x[i - HALO] : s_x[tid + HALO - 1];
    // s_xp1 = (i == N - 1 && tid == blockSize + 2) ? s_x[i + HALO] : s_x[tid + blockSize]

    // s_x = 0.f;
    // // Populate shared memory
    // if (tid < HALO) {
    //     // Left halo point for shared memory
    //     s_x[tid] = x[i - HALO];
    //     // Right halo point for shared memory
    //     s_x[tid + blockSize] = x[i + HALO];
    // }
    // // Internal shared memory points
    // s_x[tid + HALO] = x[i];

    // Synchronize threads
    __syncthreads();

    // Compute y[i]
    y[i] = -1 * s_x[tid + HALO - 1] + 2 * s_x[tid + HALO] - s_x[tid + HALO + 1];
}

int main(int argc, char * argv[]) {
    int N = 4194304;
    if (argc < 2) {
        printf("Missing Inputs");
        exit(EXIT_FAILURE);
    }

    // User defined block size
    int blockSize = atoi(argv[1]);

    // Next largest multiple of blockSize
    int numBlocks = (N + blockSize - 1) / blockSize;

    // x vector
    float * x = new float [N];
    int size_x = N * sizeof(float);

    // y vector
    float * y = new float [N];
    int size_y = N * sizeof(float);

    // Defining x_i = 1
    for (int i = 0; i < N; ++i) {
        x[i] = 1.f;
    }

    // Allocate memory and copy to the GPU
    float * d_x;
    float * d_y;
    hipMalloc((void **) &d_x, size_x);
    hipMalloc((void **) &d_y, size_y);

    // Copy memory over to the GPU
    hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size_y, hipMemcpyHostToDevice);

    stencil <<< numBlocks, blockSize >>> (N, d_y, d_x);

    // Copy memory back to the CPU
    hipMemcpy(y, d_y, size_y, hipMemcpyHostToDevice);

    // Known solution of stencil
    float *y_solution = new float[N];
    for (int i = 0; i < N; ++i) {
        y_solution[i] = 0.f;
    }

    // Initialize error to zero
    float error = 0.f;
    for (int i = 0; i < N; ++i) {
      error += fabs(y[i] - y_solution[i]);
    }
    printf("error = %f\n", error);

#if 1
  int num_trials = 10;
  hipEvent_t start, stop;
  float time;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < num_trials; ++i){
    stencil <<< numBlocks, blockSize >>> (N, d_y, d_x);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  float average_time = time / num_trials;

  printf("Time to run kernel on average: %6.6f ms.\n", average_time);
  
#endif

  // Free device memory
  hipFree(d_x);
  hipFree(d_y);

  // Free host memory
  delete[] x;
  delete[] y;
  delete[] y_solution;

  return 0;
}