#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void stencil(const int N, float * y, const float * x) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        printf("i = %d\n", i);
    }


}

int main(int argc, char * argv[]) {
    int N = 4194304;
    if (argc < 2) {
        printf("Missing Inputs");
        exit(EXIT_FAILURE);
    }

    // User defined block size
    int blockSize = atoi(argv[1]);

    // Next largest multiple of blockSize
    int numBlocks = (N + blockSize - 1) / blockSize;

    // x vector
    float * x = new float [N];
    int size_x = N * sizeof(float);

    // y vector
    float * y = new float [N];
    int size_y = N * sizeof(float);

    // Defining x_i = 1
    for (int i = 0; i < N; ++i) {
        x[i] = 1.f;
    }

    // Allocate memory and copy to the GPU
    float * d_x;
    float * d_y;
    hipMalloc((void **) &d_x, size_x);
    hipMalloc((void **) &d_y, size_y);

    // Copy memory over to the GPU
    hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size_y, hipMemcpyHostToDevice);

    stencil <<< numBlocks, blockSize >>> (N, d_y, d_x);

  // Known solution of stencil
  float * y_solution = new float[N];
  for (int i = 0; i < N; ++i) {
        y_solution[i] = 1.f;
  }

  // Initialize error to zero
  int error = 0;
  for (int i = 0; i < N; ++i) {
    error += fabs(y[i] - y_solution[i]);
  }
  printf("error = %f\n", error);

#if 1
  int num_trials = 10;
  hipEvent_t start, stop;
  float time;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < num_trials; ++i){
    stencil <<< numBlocks, blockSize >>> (N, d_y, x);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  float average_time = time / num_trials;

  printf("Time to run kernel on average: %6.6f ms.\n", average_time);
  
#endif

  return 0;
}